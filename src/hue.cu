#include "hip/hip_runtime.h"
void modify_hue(png_bytep h_image,
                int width,
                int height,
                size_t image_size,
                double hue_diff) {
    double c = cos(2 * M_PI * hue_diff);
    double s = sin(2 * M_PI * hue_diff);
    double one_third = 1.0 / 3.0;
    double sqrt_third = sqrt(one_third);

    // Matriz A compoe as operacoes de
    // conversao de RGB para HSV, mudanca de hue,
    // e conversao de HSV de volta para RGB
    // (new_r, new_g, new_b)' = A * (r, g, b)'
    // https://stackoverflow.com/questions/8507885/shift-hue-of-an-rgb-color

    double a11 = c + one_third * (1.0 - c);
    double a12 = one_third * (1.0 - c) - sqrt_third * s;
    double a13 = one_third * (1.0 - c) + sqrt_third * s;
    double a21 = a13; double a22 = a11; double a23 = a12;
    double a31 = a12; double a32 = a13; double a33 = a11;

    double A[9] = {a11, a12, a13, a21, a22, a23, a31, a32, a33};
    double *d_A;

    // Alocação de memória para matriz A na GPU
    checkErrors(hipMalloc((void **)&d_A, sizeof(double) * 9), "Alocacao da matriz A no device");

    // Copia da matriz A para a GPU
    checkErrors(hipMemcpy(d_A, A, sizeof(double) * 9, hipMemcpyHostToDevice), "Copia da matriz A para o device");

    png_bytep d_image;
    size_t d_image_size = image_size;

    // Alocação de memória para a imagem na GPU
    checkErrors(hipMalloc((void **)&d_image, d_image_size), "Alocacao da imagem no device");

    // Copia da imagem para a GPU
    checkErrors(hipMemcpy(d_image, h_image, d_image_size, hipMemcpyHostToDevice), "Copia da imagem para o device");

    // Determinar as dimensões do grid e dos blocos
    dim3 dim_block(16, 16);
    dim3 dim_grid((width + dim_block.x - 1) / dim_block.x, (height + dim_block.y - 1) / dim_block.y);

    // Chamar o kernel CUDA para processar a imagem em paralelo
    modify_hue_kernel<<<dim_grid, dim_block>>>(d_image, width, height, d_A);
    checkErrors(hipGetLastError(), "Lancamento do kernel");

    // Copia da imagem de volta para o host
    checkErrors(hipMemcpy(h_image, d_image, d_image_size, hipMemcpyDeviceToHost), "Copia da imagem para o host");

    // Liberar memória da GPU
    hipFree(d_A);
    hipFree(d_image);
}
